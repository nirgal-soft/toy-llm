#include "hip/hip_runtime.h"
#include "training.h"
#include "kernels.cuh"
#include "data_prep.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <random>

namespace training{

//helper: allocate device memory
void cuda_malloc_check(void** ptr, size_t size, const char* name){
  hipError_t err = hipMalloc(ptr, size);
  if(err != hipSuccess){
    std::cerr << "failed to allocate " << name << ": " << hipGetErrorString(err) << '\n';
    exit(1);
  }
}

void allocate_model(TrainingState* state, const ModelConfig& config){
  state->config = config;

  int num_layers = config.num_layers;
  int embed_dim = config.embed_dim;
  int vocab_size = config.vocab_size;
  int seq_len = config.seq_len;
  int batch_size = config.batch_size;
  int mlp_hidden = 4 * embed_dim;

  //allocate embeddings
  cuda_malloc_check((void**)&state->weights.token_embeddings,
                    vocab_size * embed_dim * sizeof(float), "token_embeddings");
  cuda_malloc_check((void**)&state->weights.position_embeddings,
                    seq_len * embed_dim * sizeof(float), "position_embeddings");

  //allocate per-layer arrays
  state->weights.attention_query_weights = new float*[num_layers];
  state->weights.attention_key_weights = new float*[num_layers];
  state->weights.attention_value_weights = new float*[num_layers];
  state->weights.attention_output_weights = new float*[num_layers];

  state->weights.attention_query_bias = new float*[num_layers];
  state->weights.attention_key_bias = new float*[num_layers];
  state->weights.attention_value_bias = new float*[num_layers];
  state->weights.attention_output_bias = new float*[num_layers];

  state->weights.ln1_gamma = new float*[num_layers];
  state->weights.ln1_beta = new float*[num_layers];
  state->weights.ln2_gamma = new float*[num_layers];
  state->weights.ln2_beta = new float*[num_layers];

  state->weights.mlp_fc1_weights = new float*[num_layers];
  state->weights.mlp_fc1_bias = new float*[num_layers];
  state->weights.mlp_fc2_weights = new float*[num_layers];
  state->weights.mlp_fc2_bias = new float*[num_layers];

  //allocate per-layer weights
  for(int i = 0; i < num_layers; i++){
    cuda_malloc_check((void**)&state->weights.attention_query_weights[i],
                      embed_dim * embed_dim * sizeof(float), "attn_q_weights");
    cuda_malloc_check((void**)&state->weights.attention_key_weights[i],
                      embed_dim * embed_dim * sizeof(float), "attn_k_weights");
    cuda_malloc_check((void**)&state->weights.attention_value_weights[i],
                      embed_dim * embed_dim * sizeof(float), "attn_v_weights");
    cuda_malloc_check((void**)&state->weights.attention_output_weights[i],
                      embed_dim * embed_dim * sizeof(float), "attn_out_weights");

    cuda_malloc_check((void**)&state->weights.attention_query_bias[i],
                      embed_dim * sizeof(float), "attn_q_bias");
    cuda_malloc_check((void**)&state->weights.attention_key_bias[i],
                      embed_dim * sizeof(float), "attn_k_bias");
    cuda_malloc_check((void**)&state->weights.attention_value_bias[i],
                      embed_dim * sizeof(float), "attn_v_bias");
    cuda_malloc_check((void**)&state->weights.attention_output_bias[i],
                      embed_dim * sizeof(float), "attn_out_bias");

    cuda_malloc_check((void**)&state->weights.ln1_gamma[i],
                      embed_dim * sizeof(float), "ln1_gamma");
    cuda_malloc_check((void**)&state->weights.ln1_beta[i],
                      embed_dim * sizeof(float), "ln1_beta");
    cuda_malloc_check((void**)&state->weights.ln2_gamma[i],
                      embed_dim * sizeof(float), "ln2_gamma");
    cuda_malloc_check((void**)&state->weights.ln2_beta[i],
                      embed_dim * sizeof(float), "ln2_beta");

    cuda_malloc_check((void**)&state->weights.mlp_fc1_weights[i],
                      embed_dim * mlp_hidden * sizeof(float), "mlp_fc1_weights");
    cuda_malloc_check((void**)&state->weights.mlp_fc1_bias[i],
                      mlp_hidden * sizeof(float), "mlp_fc1_bias");
    cuda_malloc_check((void**)&state->weights.mlp_fc2_weights[i],
                      mlp_hidden * embed_dim * sizeof(float), "mlp_fc2_weights");
    cuda_malloc_check((void**)&state->weights.mlp_fc2_bias[i],
                      embed_dim * sizeof(float), "mlp_fc2_bias");
  }

  //final layer
  cuda_malloc_check((void**)&state->weights.final_ln_gamma,
                    embed_dim * sizeof(float), "final_ln_gamma");
  cuda_malloc_check((void**)&state->weights.final_ln_beta,
                    embed_dim * sizeof(float), "final_ln_beta");
  cuda_malloc_check((void**)&state->weights.output_weights,
                    embed_dim * vocab_size * sizeof(float), "output_weights");

  // Allocate activation buffers
  state->activations.embedded_tokens = nullptr;
  cuda_malloc_check((void**)&state->activations.embedded_tokens,
                    batch_size * seq_len * embed_dim * sizeof(float), "embedded_tokens");

  state->activations.layer_inputs = new float*[num_layers];
  state->activations.queries = new float*[num_layers];
  state->activations.keys = new float*[num_layers];
  state->activations.values = new float*[num_layers];
  state->activations.queries_reshaped = new float*[num_layers];
  state->activations.keys_reshaped = new float*[num_layers];
  state->activations.values_reshaped = new float*[num_layers];
  state->activations.attention_scores = new float*[num_layers];
  state->activations.attention_weights = new float*[num_layers];
  state->activations.attention_output = new float*[num_layers];
  state->activations.attention_proj = new float*[num_layers];
  state->activations.post_attn = new float*[num_layers];
  state->activations.ln1_outputs = new float*[num_layers];
  state->activations.mlp_fc1 = new float*[num_layers];
  state->activations.mlp_gelu = new float*[num_layers];
  state->activations.mlp_fc2 = new float*[num_layers];
  state->activations.post_mlp = new float*[num_layers];
  state->activations.ln2_outputs = new float*[num_layers];
  state->activations.query_input = new float*[num_layers];
  state->activations.key_input = new float*[num_layers];
  state->activations.value_input = new float*[num_layers];
  state->activations.mlp_fc1_input = new float*[num_layers];

  int num_heads = config.num_heads;

  for (int i = 0; i < num_layers; i++) {
    cuda_malloc_check((void**)&state->activations.layer_inputs[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "layer_inputs");
    cuda_malloc_check((void**)&state->activations.queries[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "queries");
    cuda_malloc_check((void**)&state->activations.keys[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "keys");
    cuda_malloc_check((void**)&state->activations.values[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "values");
    cuda_malloc_check((void**)&state->activations.queries_reshaped[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "queries_reshaped");
    cuda_malloc_check((void**)&state->activations.keys_reshaped[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "keys_reshaped");
    cuda_malloc_check((void**)&state->activations.values_reshaped[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "values_reshaped");
    cuda_malloc_check((void**)&state->activations.attention_scores[i],
                      batch_size * num_heads * seq_len * seq_len * sizeof(float), "attention_scores");
    cuda_malloc_check((void**)&state->activations.attention_weights[i],
                      batch_size * num_heads * seq_len * seq_len * sizeof(float), "attention_weights");
    cuda_malloc_check((void**)&state->activations.attention_output[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "attention_output");
    cuda_malloc_check((void**)&state->activations.attention_proj[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "attention_proj");
    cuda_malloc_check((void**)&state->activations.post_attn[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "post_attn");
    cuda_malloc_check((void**)&state->activations.ln1_outputs[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "ln1_outputs");
    cuda_malloc_check((void**)&state->activations.mlp_fc1[i],
                      batch_size * seq_len * mlp_hidden * sizeof(float), "mlp_fc1");
    cuda_malloc_check((void**)&state->activations.mlp_gelu[i],
                      batch_size * seq_len * mlp_hidden * sizeof(float), "mlp_gelu");
    cuda_malloc_check((void**)&state->activations.mlp_fc2[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "mlp_fc2");
    cuda_malloc_check((void**)&state->activations.post_mlp[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "post_mlp");
    cuda_malloc_check((void**)&state->activations.ln2_outputs[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "ln2_outputs");
    cuda_malloc_check((void**)&state->activations.query_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "query_input");
    cuda_malloc_check((void**)&state->activations.key_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "key_input");
    cuda_malloc_check((void**)&state->activations.value_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "value_input");
    cuda_malloc_check((void**)&state->activations.mlp_fc1_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "mlp_fc1_input");
  }

  cuda_malloc_check((void**)&state->activations.final_ln_output,
                    batch_size * seq_len * embed_dim * sizeof(float), "final_ln_output");
  cuda_malloc_check((void**)&state->activations.logits,
                    batch_size * seq_len * vocab_size * sizeof(float), "logits");
  cuda_malloc_check((void**)&state->activations.softmax_output,
                    batch_size * seq_len * vocab_size * sizeof(float), "softmax_output");
  cuda_malloc_check((void**)&state->activations.loss,
                    sizeof(float), "loss");

  // Allocate gradient buffers (same structure as activations)
  state->gradients.token_embeddings = nullptr;
  cuda_malloc_check((void**)&state->gradients.token_embeddings,
                    vocab_size * embed_dim * sizeof(float), "grad_token_embeddings");

  // Allocate gradient weight arrays
  state->gradients.attention_query_weights = new float*[num_layers];
  state->gradients.attention_key_weights = new float*[num_layers];
  state->gradients.attention_value_weights = new float*[num_layers];
  state->gradients.attention_output_weights = new float*[num_layers];
  state->gradients.attention_query_bias = new float*[num_layers];
  state->gradients.attention_key_bias = new float*[num_layers];
  state->gradients.attention_value_bias = new float*[num_layers];
  state->gradients.attention_output_bias = new float*[num_layers];
  state->gradients.ln1_gamma = new float*[num_layers];
  state->gradients.ln1_beta = new float*[num_layers];
  state->gradients.ln2_gamma = new float*[num_layers];
  state->gradients.ln2_beta = new float*[num_layers];
  state->gradients.mlp_fc1_weights = new float*[num_layers];
  state->gradients.mlp_fc1_bias = new float*[num_layers];
  state->gradients.mlp_fc2_weights = new float*[num_layers];
  state->gradients.mlp_fc2_bias = new float*[num_layers];

  state->gradients.layer_inputs = new float*[num_layers];
  state->gradients.queries = new float*[num_layers];
  state->gradients.keys = new float*[num_layers];
  state->gradients.values = new float*[num_layers];
  state->gradients.query_input = new float*[num_layers];
  state->gradients.key_input = new float*[num_layers];
  state->gradients.value_input = new float*[num_layers];
  state->gradients.attention_scores = new float*[num_layers];
  state->gradients.attention_weights = new float*[num_layers];
  state->gradients.attention_output = new float*[num_layers];
  state->gradients.attention_proj = new float*[num_layers];
  state->gradients.post_attn = new float*[num_layers];
  state->gradients.ln1_outputs = new float*[num_layers];
  state->gradients.mlp_fc1 = new float*[num_layers];
  state->gradients.mlp_fc1_input = new float*[num_layers];
  state->gradients.mlp_gelu = new float*[num_layers];
  state->gradients.mlp_fc2 = new float*[num_layers];
  state->gradients.post_mlp = new float*[num_layers];
  state->gradients.ln2_outputs = new float*[num_layers];

  for (int i = 0; i < num_layers; i++) {
    cuda_malloc_check((void**)&state->gradients.layer_inputs[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_layer_inputs");
    cuda_malloc_check((void**)&state->gradients.queries[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_queries");
    cuda_malloc_check((void**)&state->gradients.keys[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_keys");
    cuda_malloc_check((void**)&state->gradients.values[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_values");
    cuda_malloc_check((void**)&state->gradients.query_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_query_input");
    cuda_malloc_check((void**)&state->gradients.key_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_key_input");
    cuda_malloc_check((void**)&state->gradients.value_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_value_input");
    cuda_malloc_check((void**)&state->gradients.attention_scores[i],
                      batch_size * num_heads * seq_len * seq_len * sizeof(float), "grad_attention_scores");
    cuda_malloc_check((void**)&state->gradients.attention_weights[i],
                      batch_size * num_heads * seq_len * seq_len * sizeof(float), "grad_attention_weights");
    cuda_malloc_check((void**)&state->gradients.attention_output[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_attention_output");
    cuda_malloc_check((void**)&state->gradients.attention_proj[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_attention_proj");
    cuda_malloc_check((void**)&state->gradients.post_attn[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_post_attn");
    cuda_malloc_check((void**)&state->gradients.ln1_outputs[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_ln1_outputs");
    cuda_malloc_check((void**)&state->gradients.mlp_fc1[i],
                      batch_size * seq_len * mlp_hidden * sizeof(float), "grad_mlp_fc1");
    cuda_malloc_check((void**)&state->gradients.mlp_fc1_input[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_mlp_fc1_input");
    cuda_malloc_check((void**)&state->gradients.mlp_gelu[i],
                      batch_size * seq_len * mlp_hidden * sizeof(float), "grad_mlp_gelu");
    cuda_malloc_check((void**)&state->gradients.mlp_fc2[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_mlp_fc2");
    cuda_malloc_check((void**)&state->gradients.post_mlp[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_post_mlp");
    cuda_malloc_check((void**)&state->gradients.ln2_outputs[i],
                      batch_size * seq_len * embed_dim * sizeof(float), "grad_ln2_outputs");

    // Weight gradients
    cuda_malloc_check((void**)&state->gradients.attention_query_weights[i],
                      embed_dim * embed_dim * sizeof(float), "grad_attn_q_weights");
    cuda_malloc_check((void**)&state->gradients.attention_key_weights[i],
                      embed_dim * embed_dim * sizeof(float), "grad_attn_k_weights");
    cuda_malloc_check((void**)&state->gradients.attention_value_weights[i],
                      embed_dim * embed_dim * sizeof(float), "grad_attn_v_weights");
    cuda_malloc_check((void**)&state->gradients.attention_output_weights[i],
                      embed_dim * embed_dim * sizeof(float), "grad_attn_out_weights");
    cuda_malloc_check((void**)&state->gradients.attention_query_bias[i],
                      embed_dim * sizeof(float), "grad_attn_q_bias");
    cuda_malloc_check((void**)&state->gradients.attention_key_bias[i],
                      embed_dim * sizeof(float), "grad_attn_k_bias");
    cuda_malloc_check((void**)&state->gradients.attention_value_bias[i],
                      embed_dim * sizeof(float), "grad_attn_v_bias");
    cuda_malloc_check((void**)&state->gradients.attention_output_bias[i],
                      embed_dim * sizeof(float), "grad_attn_out_bias");
    cuda_malloc_check((void**)&state->gradients.ln1_gamma[i],
                      embed_dim * sizeof(float), "grad_ln1_gamma");
    cuda_malloc_check((void**)&state->gradients.ln1_beta[i],
                      embed_dim * sizeof(float), "grad_ln1_beta");
    cuda_malloc_check((void**)&state->gradients.ln2_gamma[i],
                      embed_dim * sizeof(float), "grad_ln2_gamma");
    cuda_malloc_check((void**)&state->gradients.ln2_beta[i],
                      embed_dim * sizeof(float), "grad_ln2_beta");
    cuda_malloc_check((void**)&state->gradients.mlp_fc1_weights[i],
                      embed_dim * mlp_hidden * sizeof(float), "grad_mlp_fc1_weights");
    cuda_malloc_check((void**)&state->gradients.mlp_fc1_bias[i],
                      mlp_hidden * sizeof(float), "grad_mlp_fc1_bias");
    cuda_malloc_check((void**)&state->gradients.mlp_fc2_weights[i],
                      mlp_hidden * embed_dim * sizeof(float), "grad_mlp_fc2_weights");
    cuda_malloc_check((void**)&state->gradients.mlp_fc2_bias[i],
                      embed_dim * sizeof(float), "grad_mlp_fc2_bias");
  }

  // Final layer weight gradients (MISSING - add this!)
  cuda_malloc_check((void**)&state->gradients.position_embeddings,
                    seq_len * embed_dim * sizeof(float), "grad_position_embeddings");
  cuda_malloc_check((void**)&state->gradients.final_ln_gamma,
                    embed_dim * sizeof(float), "grad_final_ln_gamma");
  cuda_malloc_check((void**)&state->gradients.final_ln_beta,
                    embed_dim * sizeof(float), "grad_final_ln_beta");
  cuda_malloc_check((void**)&state->gradients.output_weights,
                    embed_dim * vocab_size * sizeof(float), "grad_output_weights");

  cuda_malloc_check((void**)&state->gradients.final_ln_output,
                    batch_size * seq_len * embed_dim * sizeof(float), "grad_final_ln_output");
  cuda_malloc_check((void**)&state->gradients.logits,
                    batch_size * seq_len * vocab_size * sizeof(float), "grad_logits");

  // Allocate optimizer state (momentum and velocity)
  state->optimizer.timestep = 0;

  // Embeddings momentum and velocity
  cuda_malloc_check((void**)&state->optimizer.momentum.token_embeddings,
                    vocab_size * embed_dim * sizeof(float), "momentum_token_embeddings");
  cuda_malloc_check((void**)&state->optimizer.velocity.token_embeddings,
                    vocab_size * embed_dim * sizeof(float), "velocity_token_embeddings");
  hipMemset(state->optimizer.momentum.token_embeddings, 0, vocab_size * embed_dim * sizeof(float));
  hipMemset(state->optimizer.velocity.token_embeddings, 0, vocab_size * embed_dim * sizeof(float));

  cuda_malloc_check((void**)&state->optimizer.momentum.position_embeddings,
                   seq_len * embed_dim * sizeof(float), "momentum_position_embeddings");
  cuda_malloc_check((void**)&state->optimizer.velocity.position_embeddings,
                    seq_len * embed_dim * sizeof(float), "velocity_position_embeddings");
  hipMemset(state->optimizer.momentum.position_embeddings, 0, seq_len * embed_dim * sizeof(float));
  hipMemset(state->optimizer.velocity.position_embeddings, 0, seq_len * embed_dim * sizeof(float));

  // Allocate per-layer momentum and velocity arrays
  state->optimizer.momentum.attention_query_weights = new float*[num_layers];
  state->optimizer.momentum.attention_key_weights = new float*[num_layers];
  state->optimizer.momentum.attention_value_weights = new float*[num_layers];
  state->optimizer.momentum.attention_output_weights = new float*[num_layers];
  state->optimizer.momentum.attention_query_bias = new float*[num_layers];
  state->optimizer.momentum.attention_key_bias = new float*[num_layers];
  state->optimizer.momentum.attention_value_bias = new float*[num_layers];
  state->optimizer.momentum.attention_output_bias = new float*[num_layers];
  state->optimizer.momentum.ln1_gamma = new float*[num_layers];
  state->optimizer.momentum.ln1_beta = new float*[num_layers];
  state->optimizer.momentum.ln2_gamma = new float*[num_layers];
  state->optimizer.momentum.ln2_beta = new float*[num_layers];
  state->optimizer.momentum.mlp_fc1_weights = new float*[num_layers];
  state->optimizer.momentum.mlp_fc1_bias = new float*[num_layers];
  state->optimizer.momentum.mlp_fc2_weights = new float*[num_layers];
  state->optimizer.momentum.mlp_fc2_bias = new float*[num_layers];

  state->optimizer.velocity.attention_query_weights = new float*[num_layers];
  state->optimizer.velocity.attention_key_weights = new float*[num_layers];
  state->optimizer.velocity.attention_value_weights = new float*[num_layers];
  state->optimizer.velocity.attention_output_weights = new float*[num_layers];
  state->optimizer.velocity.attention_query_bias = new float*[num_layers];
  state->optimizer.velocity.attention_key_bias = new float*[num_layers];
  state->optimizer.velocity.attention_value_bias = new float*[num_layers];
  state->optimizer.velocity.attention_output_bias = new float*[num_layers];
  state->optimizer.velocity.ln1_gamma = new float*[num_layers];
  state->optimizer.velocity.ln1_beta = new float*[num_layers];
  state->optimizer.velocity.ln2_gamma = new float*[num_layers];
  state->optimizer.velocity.ln2_beta = new float*[num_layers];
  state->optimizer.velocity.mlp_fc1_weights = new float*[num_layers];
  state->optimizer.velocity.mlp_fc1_bias = new float*[num_layers];
  state->optimizer.velocity.mlp_fc2_weights = new float*[num_layers];
  state->optimizer.velocity.mlp_fc2_bias = new float*[num_layers];

  for (int i = 0; i < num_layers; i++) {
    // Attention weights momentum and velocity
    cuda_malloc_check((void**)&state->optimizer.momentum.attention_query_weights[i],
                      embed_dim * embed_dim * sizeof(float), "momentum_attn_q_weights");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_query_weights[i],
                      embed_dim * embed_dim * sizeof(float), "velocity_attn_q_weights");
    hipMemset(state->optimizer.momentum.attention_query_weights[i], 0, embed_dim * embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_query_weights[i], 0, embed_dim * embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.attention_key_weights[i],
                      embed_dim * embed_dim * sizeof(float), "momentum_attn_k_weights");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_key_weights[i],
                      embed_dim * embed_dim * sizeof(float), "velocity_attn_k_weights");
    hipMemset(state->optimizer.momentum.attention_key_weights[i], 0, embed_dim * embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_key_weights[i], 0, embed_dim * embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.attention_value_weights[i],
                      embed_dim * embed_dim * sizeof(float), "momentum_attn_v_weights");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_value_weights[i],
                      embed_dim * embed_dim * sizeof(float), "velocity_attn_v_weights");
    hipMemset(state->optimizer.momentum.attention_value_weights[i], 0, embed_dim * embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_value_weights[i], 0, embed_dim * embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.attention_output_weights[i],
                      embed_dim * embed_dim * sizeof(float), "momentum_attn_out_weights");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_output_weights[i],
                      embed_dim * embed_dim * sizeof(float), "velocity_attn_out_weights");
    hipMemset(state->optimizer.momentum.attention_output_weights[i], 0, embed_dim * embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_output_weights[i], 0, embed_dim * embed_dim * sizeof(float));

    // Attention biases momentum and velocity
    cuda_malloc_check((void**)&state->optimizer.momentum.attention_query_bias[i],
                      embed_dim * sizeof(float), "momentum_attn_q_bias");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_query_bias[i],
                      embed_dim * sizeof(float), "velocity_attn_q_bias");
    hipMemset(state->optimizer.momentum.attention_query_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_query_bias[i], 0, embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.attention_key_bias[i],
                      embed_dim * sizeof(float), "momentum_attn_k_bias");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_key_bias[i],
                      embed_dim * sizeof(float), "velocity_attn_k_bias");
    hipMemset(state->optimizer.momentum.attention_key_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_key_bias[i], 0, embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.attention_value_bias[i],
                      embed_dim * sizeof(float), "momentum_attn_v_bias");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_value_bias[i],
                      embed_dim * sizeof(float), "velocity_attn_v_bias");
    hipMemset(state->optimizer.momentum.attention_value_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_value_bias[i], 0, embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.attention_output_bias[i],
                      embed_dim * sizeof(float), "momentum_attn_out_bias");
    cuda_malloc_check((void**)&state->optimizer.velocity.attention_output_bias[i],
                      embed_dim * sizeof(float), "velocity_attn_out_bias");
    hipMemset(state->optimizer.momentum.attention_output_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.attention_output_bias[i], 0, embed_dim * sizeof(float));

    // Layer norm momentum and velocity
    cuda_malloc_check((void**)&state->optimizer.momentum.ln1_gamma[i],
                      embed_dim * sizeof(float), "momentum_ln1_gamma");
    cuda_malloc_check((void**)&state->optimizer.velocity.ln1_gamma[i],
                      embed_dim * sizeof(float), "velocity_ln1_gamma");
    hipMemset(state->optimizer.momentum.ln1_gamma[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.ln1_gamma[i], 0, embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.ln1_beta[i],
                      embed_dim * sizeof(float), "momentum_ln1_beta");
    cuda_malloc_check((void**)&state->optimizer.velocity.ln1_beta[i],
                      embed_dim * sizeof(float), "velocity_ln1_beta");
    hipMemset(state->optimizer.momentum.ln1_beta[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.ln1_beta[i], 0, embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.ln2_gamma[i],
                      embed_dim * sizeof(float), "momentum_ln2_gamma");
    cuda_malloc_check((void**)&state->optimizer.velocity.ln2_gamma[i],
                      embed_dim * sizeof(float), "velocity_ln2_gamma");
    hipMemset(state->optimizer.momentum.ln2_gamma[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.ln2_gamma[i], 0, embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.ln2_beta[i],
                      embed_dim * sizeof(float), "momentum_ln2_beta");
    cuda_malloc_check((void**)&state->optimizer.velocity.ln2_beta[i],
                      embed_dim * sizeof(float), "velocity_ln2_beta");
    hipMemset(state->optimizer.momentum.ln2_beta[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.ln2_beta[i], 0, embed_dim * sizeof(float));

    // MLP weights momentum and velocity
    cuda_malloc_check((void**)&state->optimizer.momentum.mlp_fc1_weights[i],
                      embed_dim * mlp_hidden * sizeof(float), "momentum_mlp_fc1_weights");
    cuda_malloc_check((void**)&state->optimizer.velocity.mlp_fc1_weights[i],
                      embed_dim * mlp_hidden * sizeof(float), "velocity_mlp_fc1_weights");
    hipMemset(state->optimizer.momentum.mlp_fc1_weights[i], 0, embed_dim * mlp_hidden * sizeof(float));
    hipMemset(state->optimizer.velocity.mlp_fc1_weights[i], 0, embed_dim * mlp_hidden * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.mlp_fc1_bias[i],
                      mlp_hidden * sizeof(float), "momentum_mlp_fc1_bias");
    cuda_malloc_check((void**)&state->optimizer.velocity.mlp_fc1_bias[i],
                      mlp_hidden * sizeof(float), "velocity_mlp_fc1_bias");
    hipMemset(state->optimizer.momentum.mlp_fc1_bias[i], 0, mlp_hidden * sizeof(float));
    hipMemset(state->optimizer.velocity.mlp_fc1_bias[i], 0, mlp_hidden * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.mlp_fc2_weights[i],
                      mlp_hidden * embed_dim * sizeof(float), "momentum_mlp_fc2_weights");
    cuda_malloc_check((void**)&state->optimizer.velocity.mlp_fc2_weights[i],
                      mlp_hidden * embed_dim * sizeof(float), "velocity_mlp_fc2_weights");
    hipMemset(state->optimizer.momentum.mlp_fc2_weights[i], 0, mlp_hidden * embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.mlp_fc2_weights[i], 0, mlp_hidden * embed_dim * sizeof(float));

    cuda_malloc_check((void**)&state->optimizer.momentum.mlp_fc2_bias[i],
                      embed_dim * sizeof(float), "momentum_mlp_fc2_bias");
    cuda_malloc_check((void**)&state->optimizer.velocity.mlp_fc2_bias[i],
                      embed_dim * sizeof(float), "velocity_mlp_fc2_bias");
    hipMemset(state->optimizer.momentum.mlp_fc2_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->optimizer.velocity.mlp_fc2_bias[i], 0, embed_dim * sizeof(float));
  }

  // Final layer momentum and velocity
  cuda_malloc_check((void**)&state->optimizer.momentum.final_ln_gamma,
                    embed_dim * sizeof(float), "momentum_final_ln_gamma");
  cuda_malloc_check((void**)&state->optimizer.velocity.final_ln_gamma,
                    embed_dim * sizeof(float), "velocity_final_ln_gamma");
  hipMemset(state->optimizer.momentum.final_ln_gamma, 0, embed_dim * sizeof(float));
  hipMemset(state->optimizer.velocity.final_ln_gamma, 0, embed_dim * sizeof(float));

  cuda_malloc_check((void**)&state->optimizer.momentum.final_ln_beta,
                    embed_dim * sizeof(float), "momentum_final_ln_beta");
  cuda_malloc_check((void**)&state->optimizer.velocity.final_ln_beta,
                    embed_dim * sizeof(float), "velocity_final_ln_beta");
  hipMemset(state->optimizer.momentum.final_ln_beta, 0, embed_dim * sizeof(float));
  hipMemset(state->optimizer.velocity.final_ln_beta, 0, embed_dim * sizeof(float));

  cuda_malloc_check((void**)&state->optimizer.momentum.output_weights,
                    embed_dim * vocab_size * sizeof(float), "momentum_output_weights");
  cuda_malloc_check((void**)&state->optimizer.velocity.output_weights,
                    embed_dim * vocab_size * sizeof(float), "velocity_output_weights");
  hipMemset(state->optimizer.momentum.output_weights, 0, embed_dim * vocab_size * sizeof(float));
  hipMemset(state->optimizer.velocity.output_weights, 0, embed_dim * vocab_size * sizeof(float));

  std::cout << "model allocated successfully" << '\n';
  std::cout << "total params: ~" <<
    (vocab_size * embed_dim +
     num_layers * (4 * embed_dim * embed_dim + 8 * embed_dim +
                  embed_dim * mlp_hidden * 2 + mlp_hidden + embed_dim) +
     embed_dim * vocab_size) / 1000000.0f << "M" << '\n';
}

void initialize_weights(TrainingState* state){
  //xavier/glorot init
  std::random_device rd;
  std::mt19937 gen(rd());

  auto init_matrix = [&](float* weights, int rows, int cols){
    float std_dev = sqrtf(2.0f / (rows+cols));
    std::normal_distribution<float> dist(0.0f, std_dev);

    std::vector<float> h_weights(rows * cols);
    for(int i = 0; i < rows * cols; i++){
      h_weights[i] = dist(gen);
    }
    hipMemcpy(weights, h_weights.data(), rows*cols*sizeof(float), hipMemcpyHostToDevice);
  };

  int embed_dim = state->config.embed_dim;
  int vocab_size = state->config.vocab_size;
  int seq_len = state->config.seq_len;
  int num_layers = state->config.num_layers;
  int mlp_hidden = 4 * embed_dim;

  //init embeddings
  init_matrix(state->weights.token_embeddings, vocab_size, embed_dim);
  init_matrix(state->weights.position_embeddings, seq_len, embed_dim);

  std::vector<float> ones(embed_dim, 1.0f);
  //init layer weights
  for(int i = 0; i < num_layers; i++){
    init_matrix(state->weights.attention_query_weights[i], embed_dim, embed_dim);
    init_matrix(state->weights.attention_key_weights[i], embed_dim, embed_dim);
    init_matrix(state->weights.attention_value_weights[i], embed_dim, embed_dim);
    init_matrix(state->weights.attention_output_weights[i], embed_dim, embed_dim);

    hipMemset(state->weights.attention_query_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->weights.attention_key_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->weights.attention_value_bias[i], 0, embed_dim * sizeof(float));
    hipMemset(state->weights.attention_output_bias[i], 0, embed_dim * sizeof(float));

    //layer norm: gamma=1, beta=0
    hipMemcpy(state->weights.ln1_gamma[i], ones.data(), embed_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(state->weights.ln2_gamma[i], ones.data(), embed_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(state->weights.ln1_beta[i], 0, embed_dim * sizeof(float));
    hipMemset(state->weights.ln2_beta[i], 0, embed_dim * sizeof(float));

    init_matrix(state->weights.mlp_fc1_weights[i], embed_dim, mlp_hidden);
    hipMemset(state->weights.mlp_fc1_bias[i], 0, mlp_hidden * sizeof(float));
    init_matrix(state->weights.mlp_fc2_weights[i], mlp_hidden, embed_dim);
    hipMemset(state->weights.mlp_fc2_bias[i], 0, embed_dim * sizeof(float));
  }

  //final layer
  hipMemcpy(state->weights.final_ln_gamma, ones.data(), embed_dim*sizeof(float), hipMemcpyHostToDevice);
  hipMemset(state->weights.final_ln_beta, 0, embed_dim * sizeof(float));
  init_matrix(state->weights.output_weights, embed_dim, vocab_size);
  
  std::cout << "weights initialized" << '\n';
}

float compute_loss(TrainingState* state, int* target_ids, int batch_size, int seq_len){
  //reset loss to zero
  hipMemset(state->activations.loss, 0, sizeof(float));

  //compute cross-entropy loss
  int total_predictions = batch_size * seq_len;
  int block_size = 256;
  int grid_size = (total_predictions + block_size - 1)/block_size;

  cross_entropy_loss<<<grid_size, block_size>>>(
    state->activations.logits,
    target_ids,
    state->activations.loss,
    batch_size, seq_len, state->config.vocab_size
  );

  //copy loss back to host
  float h_loss;
  hipMemcpy(&h_loss, state->activations.loss, sizeof(float), hipMemcpyDeviceToHost);

  return h_loss;
}

void clip_gradients(TrainingState* state, float max_norm) {
  int embed_dim = state->config.embed_dim;
  int vocab_size = state->config.vocab_size;
  int seq_len = state->config.seq_len;
  int num_layers = state->config.num_layers;
  int mlp_hidden = 4 * embed_dim;

  // Allocate device memory for total squared norm
  float* d_total_squared_norm;
  hipMalloc(&d_total_squared_norm, sizeof(float));
  hipMemset(d_total_squared_norm, 0, sizeof(float));
  
  int block_size = 256;
  
  // === Compute global norm across ALL gradients ===
  
  // Embeddings
  compute_squared_norm_kernel<<<(vocab_size * embed_dim + block_size - 1) / block_size, block_size>>>(
    state->gradients.token_embeddings, vocab_size * embed_dim, d_total_squared_norm
  );
  compute_squared_norm_kernel<<<(seq_len * embed_dim + block_size - 1) / block_size, block_size>>>(
    state->gradients.position_embeddings, seq_len * embed_dim, d_total_squared_norm
  );
  
  // Per-layer gradients
  for (int i = 0; i < num_layers; i++) {
    // Attention weights (embed_dim * embed_dim each)
    compute_squared_norm_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_query_weights[i], embed_dim * embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_key_weights[i], embed_dim * embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_value_weights[i], embed_dim * embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_output_weights[i], embed_dim * embed_dim, d_total_squared_norm
    );
    
    // Attention biases (embed_dim each)
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_query_bias[i], embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_key_bias[i], embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_value_bias[i], embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_output_bias[i], embed_dim, d_total_squared_norm
    );
    
    // Layer norms (embed_dim each)
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln1_gamma[i], embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln1_beta[i], embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln2_gamma[i], embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln2_beta[i], embed_dim, d_total_squared_norm
    );
    
    // MLP weights
    compute_squared_norm_kernel<<<(embed_dim * mlp_hidden + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc1_weights[i], embed_dim * mlp_hidden, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(mlp_hidden + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc1_bias[i], mlp_hidden, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(mlp_hidden * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc2_weights[i], mlp_hidden * embed_dim, d_total_squared_norm
    );
    compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc2_bias[i], embed_dim, d_total_squared_norm
    );
  }
  
  // Final layer
  compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
    state->gradients.final_ln_gamma, embed_dim, d_total_squared_norm
  );
  compute_squared_norm_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
    state->gradients.final_ln_beta, embed_dim, d_total_squared_norm
  );
  compute_squared_norm_kernel<<<(embed_dim * vocab_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.output_weights, embed_dim * vocab_size, d_total_squared_norm
  );
  
  // Copy total squared norm back to host
  float h_total_squared_norm;
  hipMemcpy(&h_total_squared_norm, d_total_squared_norm, sizeof(float), hipMemcpyDeviceToHost);
  
  // Compute total norm and scale factor
  float total_norm = sqrtf(h_total_squared_norm);

  // std::cout << "Gradient norm: " << total_norm << " (max_norm=" << max_norm << ")";

  // float debug_scale = (total_norm > max_norm) ? (max_norm / total_norm) : 1.0f;
  // if (debug_scale < 1.0f) {
  //   std::cout << " -> CLIPPING with scale=" << debug_scale << std::endl;
  // } else {
  //   std::cout << " -> no clipping needed" << std::endl;
  // }

  float scale = (total_norm > max_norm) ? (max_norm / total_norm) : 1.0f;
  
  // Only scale if necessary
  if (scale < 1.0f) {
    // Scale all gradients by the computed factor
    scale_gradients_kernel<<<(vocab_size * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.token_embeddings, vocab_size * embed_dim, scale
    );
    scale_gradients_kernel<<<(seq_len * embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.position_embeddings, seq_len * embed_dim, scale
    );
    
    for (int i = 0; i < num_layers; i++) {
      scale_gradients_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_query_weights[i], embed_dim * embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_key_weights[i], embed_dim * embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_value_weights[i], embed_dim * embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim * embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_output_weights[i], embed_dim * embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_query_bias[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_key_bias[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_value_bias[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.attention_output_bias[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.ln1_gamma[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.ln1_beta[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.ln2_gamma[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.ln2_beta[i], embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim * mlp_hidden + block_size - 1) / block_size, block_size>>>(
        state->gradients.mlp_fc1_weights[i], embed_dim * mlp_hidden, scale
      );
      scale_gradients_kernel<<<(mlp_hidden + block_size - 1) / block_size, block_size>>>(
        state->gradients.mlp_fc1_bias[i], mlp_hidden, scale
      );
      scale_gradients_kernel<<<(mlp_hidden * embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.mlp_fc2_weights[i], mlp_hidden * embed_dim, scale
      );
      scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
        state->gradients.mlp_fc2_bias[i], embed_dim, scale
      );
    }
    
    scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.final_ln_gamma, embed_dim, scale
    );
    scale_gradients_kernel<<<(embed_dim + block_size - 1) / block_size, block_size>>>(
      state->gradients.final_ln_beta, embed_dim, scale
    );
    scale_gradients_kernel<<<(embed_dim * vocab_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.output_weights, embed_dim * vocab_size, scale
    );
  }

  // Log gradient norm for debugging
  float h_grad_norm = sqrtf(h_total_squared_norm);
  static int print_counter = 0;
  if(print_counter++ % 100 == 0) {
    std::cout << " grad_norm: " << h_grad_norm;
  }

  hipFree(d_total_squared_norm);
  hipDeviceSynchronize();
}

void zero_gradients(TrainingState* state){
  int embed_dim = state->config.embed_dim;
  int vocab_size = state->config.vocab_size;
  int num_layers = state->config.num_layers;
  int num_heads = state->config.num_heads;
  int batch_size = state->config.batch_size;
  int seq_len = state->config.seq_len;
  int mlp_hidden = 4 * embed_dim;

  // Zero embedding gradients
  int block_size = 256;
  int grad_size = vocab_size * embed_dim;
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.token_embeddings, grad_size
  );

  grad_size = seq_len * embed_dim;
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.position_embeddings, grad_size
  );

  // Zero per-layer gradients
  for (int i = 0; i < num_layers; i++) {
    // Attention weight gradients
    grad_size = embed_dim * embed_dim;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_query_weights[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_key_weights[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_value_weights[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_output_weights[i], grad_size
    );

    // Attention bias gradients
    grad_size = embed_dim;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_query_bias[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_key_bias[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_value_bias[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_output_bias[i], grad_size
    );

    // Layer norm gradients
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln1_gamma[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln1_beta[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln2_gamma[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln2_beta[i], grad_size
    );

    // MLP weight gradients
    grad_size = embed_dim * mlp_hidden;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc1_weights[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc2_weights[i], grad_size
    );

    // MLP bias gradients
    grad_size = mlp_hidden;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc1_bias[i], grad_size
    );
    grad_size = embed_dim;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc2_bias[i], grad_size
    );

    grad_size = batch_size * seq_len * embed_dim;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.layer_inputs[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.queries[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.keys[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.values[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.query_input[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.key_input[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.value_input[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_output[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_proj[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.post_attn[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln1_outputs[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc1_input[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.post_mlp[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.ln2_outputs[i], grad_size
    );

    grad_size = batch_size * num_heads * seq_len * seq_len;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_scores[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.attention_weights[i], grad_size
    );

    grad_size = batch_size * seq_len * mlp_hidden;
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc1[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_gelu[i], grad_size
    );
    zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
      state->gradients.mlp_fc2[i], grad_size
    );
  }

  grad_size = batch_size * seq_len * embed_dim;
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.final_ln_output, grad_size
  );

  grad_size = batch_size * seq_len * vocab_size;
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.logits, grad_size
  );

  // Zero final layer gradients
  grad_size = embed_dim;
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.final_ln_gamma, grad_size
  );
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.final_ln_beta, grad_size
  );

  grad_size = embed_dim * vocab_size;
  zero_gradients_kernel<<<(grad_size + block_size - 1) / block_size, block_size>>>(
    state->gradients.output_weights, grad_size
  );

  hipDeviceSynchronize();
};

void train_model(const std::string& token_ids_path, const ModelConfig& config,
                 int num_epochs, float learning_rate) {
  std::cout << "=== Starting Training ===" << std::endl;

  // Load training data
  auto token_ids = data_prep::load_token_ids(token_ids_path);
  std::cout << "Loaded " << token_ids.size() << " tokens" << std::endl;
  std::cout << "First 20 loaded tokens: ";
  for (int i = 0; i < 20; i++) {
    std::cout << token_ids[i] << " ";
  }
  std::cout << std::endl;

  // Create batches
  auto batches = data_prep::create_training_batches(token_ids, config.batch_size, config.seq_len);
  std::cout << "Created " << batches.size() << " training batches" << std::endl;

  if (batches.empty()) {
    std::cerr << "No training batches created!" << std::endl;
    return;
  }

  // Allocate and initialize model
  TrainingState state;
  allocate_model(&state, config);
  initialize_weights(&state);

  // Verify weights aren't zero
  std::vector<float> test_weights(100);
  hipMemcpy(test_weights.data(), state.weights.token_embeddings, 
             100 * sizeof(float), hipMemcpyDeviceToHost);
  std::cout << "Sample weights after init: ";
  for (int i = 0; i < 10; i++) {
    std::cout << test_weights[i] << " ";
  }
  std::cout << std::endl;

  // Allocate device memory for batch data
  int* d_input_tokens;
  int* d_target_tokens;
  hipMalloc(&d_input_tokens, config.batch_size * config.seq_len * sizeof(int));
  hipMalloc(&d_target_tokens, config.batch_size * config.seq_len * sizeof(int));

  // Adam hyperparameters
  float beta1 = 0.9f;
  float beta2 = 0.999f;
  float epsilon = 1e-7f;

  // Training loop
  for (int epoch = 0; epoch < num_epochs; epoch++) {
    std::cout << "\n=== Epoch " << (epoch + 1) << "/" << num_epochs << " ===" << std::endl;

    float epoch_loss = 0.0f;
    int num_batches_processed = 0;

    for (size_t batch_idx = 0; batch_idx < batches.size(); batch_idx++) {
      auto& batch = batches[batch_idx];
      int actual_batch_size = batch.input_sequences.size();

      // Skip incomplete batches
      if (actual_batch_size != config.batch_size) {
        continue;
      }

      if(batch_idx == 0){
        std::cout << "batch 0 raw data - first sequence first 10 tokens: ";
        for(int i = 0; i < 10; i++){
          std::cout << batch.input_sequences[0][i] << " ";
        }
        std::cout << '\n';
      }

      // Flatten batch data to contiguous arrays
      std::vector<int> h_input_flat(actual_batch_size * config.seq_len);
      std::vector<int> h_target_flat(actual_batch_size * config.seq_len);

      for (int b = 0; b < actual_batch_size; b++) {
        for (int s = 0; s < config.seq_len; s++) {
          h_input_flat[b * config.seq_len + s] = batch.input_sequences[b][s];
          h_target_flat[b * config.seq_len + s] = batch.target_sequences[b][s];
        }
      }

      if (batch_idx == 0) {
        std::cout << "after flattening - first 10 tokens: ";
        for (int i = 0; i < 10; i++) {
          std::cout << h_input_flat[i] << " ";
        }
        std::cout << '\n';
      }

      // Copy batch to device
      hipMemcpy(d_input_tokens, h_input_flat.data(),
                 actual_batch_size * config.seq_len * sizeof(int),
                 hipMemcpyHostToDevice);

      if (batch_idx == 0) {
        std::vector<int> verify_copy(10);
        hipMemcpy(verify_copy.data(), d_input_tokens, 10 * sizeof(int), hipMemcpyDeviceToHost);
        std::cout << "After hipMemcpy to device - first 10 tokens: ";
        for (int i = 0; i < 10; i++) {
          std::cout << verify_copy[i] << " ";
        }
        std::cout << std::endl;
      }

      hipMemcpy(d_target_tokens, h_target_flat.data(),
                 actual_batch_size * config.seq_len * sizeof(int),
                 hipMemcpyHostToDevice);

      // Zero gradients
      zero_gradients(&state);

      hipDeviceSynchronize();
      if (batch_idx == 0) {
        std::vector<int> check_before_forward(10);
        hipMemcpy(check_before_forward.data(), d_input_tokens, 10 * sizeof(int), hipMemcpyDeviceToHost);
        std::cout << "Right before forward_pass - first 10 tokens: ";
        for (int i = 0; i < 10; i++) {
          std::cout << check_before_forward[i] << " ";
        }
        std::cout << std::endl;
      }

      int total_steps = batches.size();
      int warmup_steps = 500;  // Standard warmup
      float current_lr = get_learning_rate(batch_idx, warmup_steps, total_steps, learning_rate);

      if(batch_idx % 100 == 0){
        std::cout << " current lr: " << current_lr << '\n';
      }

      // Forward pass
      forward_pass(&state, d_input_tokens, actual_batch_size, config.seq_len);

      // Check for CUDA errors
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
        std::cerr << "CUDA error after forward pass: " << hipGetErrorString(err) << std::endl;
        exit(1);
      }
      hipDeviceSynchronize();
      err = hipGetLastError();
      if (err != hipSuccess) {
        std::cerr << "CUDA error after synchronize: " << hipGetErrorString(err) << std::endl;
        exit(1);
      }

      // Add this AFTER forward_pass, BEFORE compute_loss
      // if (batch_idx % 100 == 0) {
      //   std::vector<float> sample_acts(100);
      //   hipMemcpy(sample_acts.data(), state.activations.queries[0], 
      //              100 * sizeof(float), hipMemcpyDeviceToHost);

      //   float min_act = *std::min(sample_acts.begin(), sample_acts.end());
      //   float max_act = *std::max(sample_acts.begin(), sample_acts.end());
      //   std::cout << "  Forward activations (queries): [" << min_act << ", " << max_act << "]" << std::endl;
      // }

      // Compute loss
      float batch_loss = compute_loss(&state, d_target_tokens, actual_batch_size, config.seq_len);

      // Add diagnostics
      if (batch_idx == 0) {
        int vocab_size = config.vocab_size;
        // Copy a few logits to check
        std::vector<float> sample_logits(vocab_size);
        hipMemcpy(sample_logits.data(), state.activations.logits, 
                   vocab_size * sizeof(float), hipMemcpyDeviceToHost);
        
        std::cout << "First logit values: ";
        for (int i = 0; i < std::min(10, vocab_size); i++) {
          std::cout << sample_logits[i] << " ";
        }
        std::cout << std::endl;
        
        std::cout << "First target token: " << h_target_flat[0] << std::endl;
        std::cout << "Batch loss: " << batch_loss << std::endl;
        std::cout << "Vocab size: " << vocab_size << std::endl;
      }

      epoch_loss += batch_loss;
      num_batches_processed++;

      // Backward pass
      backward_pass(&state, d_input_tokens, d_target_tokens, actual_batch_size, config.seq_len);

      //clip gradients
      clip_gradients(&state, 5.0f);

     

      // if (batch_idx % 10 == 0) {
      //   // Check gradients for tokens that actually appear in the batch
      //   std::vector<float> token_29_grad(128);
      //   hipMemcpy(token_29_grad.data(), 
      //              state.gradients.token_embeddings + 29 * state.config.embed_dim,
      //              128 * sizeof(float), hipMemcpyDeviceToHost);

      //   float grad_norm = 0;
      //   for (float g : token_29_grad) {
      //     grad_norm += g * g;
      //   }
      //   grad_norm = sqrt(grad_norm);

      //   std::cout << "  Token 29 gradient norm: " << grad_norm << std::endl;

      //   // Check if token 29's embedding is updating
      //   std::vector<float> token_29_weights(128);
      //   hipMemcpy(token_29_weights.data(),
      //              state.weights.token_embeddings + 29 * state.config.embed_dim,
      //              128 * sizeof(float), hipMemcpyDeviceToHost);

      //   float weight_norm = 0;
      //   for (float w : token_29_weights) {
      //     weight_norm += w * w;
      //   }
      //   weight_norm = sqrt(weight_norm);

      //   std::cout << "  Token 29 weight norm: " << weight_norm << std::endl;
      // }

      // if (batch_idx % 100 == 0) {
      //   std::vector<float> attn_grads(100), mlp_grads(100);
      //   hipMemcpy(attn_grads.data(), state.gradients.attention_query_weights[0], 
      //              100 * sizeof(float), hipMemcpyDeviceToHost);
      //   hipMemcpy(mlp_grads.data(), state.gradients.mlp_fc1_weights[0], 
      //              100 * sizeof(float), hipMemcpyDeviceToHost);

      //   auto norm = [](const std::vector<float>& v) {
      //     float sum = 0;
      //     for (float x : v) sum += x*x;
      //     return sqrt(sum);
      //   };

      //   std::cout << "  Attn grad norm: " << norm(attn_grads) 
      //     << ", MLP grad norm: " << norm(mlp_grads) << std::endl;
      // }

      // Optimizer step
      optimizer_step(&state, current_lr, beta1, beta2, epsilon);

      // Print progress
      if (batch_idx % 10 == 0) {
        std::cout << "Batch " << batch_idx << "/" << batches.size() 
          << " - Loss: " << batch_loss << std::endl;
      }
      if(batch_idx > 0 && batch_idx % 500 == 0){
        char checkpoint_name[256];
        sprintf(checkpoint_name, "./data/checkpoints/model_batch_%d.bin", num_batches_processed);
        // std::string checkpoint_path = "./data/checkpoints/model.bin";
        save_checkpoint(&state, checkpoint_name, epoch + 1, 1.0f);
      }
    }

    float avg_loss = epoch_loss / num_batches_processed;
    std::cout << "Epoch " << (epoch + 1) << " complete - Average loss: " << avg_loss << std::endl;
  }

  // Cleanup
  hipFree(d_input_tokens);
  hipFree(d_target_tokens);
  free_model(&state);

  std::cout << "\n=== Training Complete ===" << std::endl;
}

void free_model(TrainingState* state) {
  int num_layers = state->config.num_layers;

  // Free embeddings
  hipFree(state->weights.token_embeddings);
  hipFree(state->weights.position_embeddings);

  // Free per-layer weights, gradients, activations, and optimizer state
  for (int i = 0; i < num_layers; i++) {
    // Weights
    hipFree(state->weights.attention_query_weights[i]);
    hipFree(state->weights.attention_key_weights[i]);
    hipFree(state->weights.attention_value_weights[i]);
    hipFree(state->weights.attention_output_weights[i]);
    hipFree(state->weights.attention_query_bias[i]);
    hipFree(state->weights.attention_key_bias[i]);
    hipFree(state->weights.attention_value_bias[i]);
    hipFree(state->weights.attention_output_bias[i]);
    hipFree(state->weights.ln1_gamma[i]);
    hipFree(state->weights.ln1_beta[i]);
    hipFree(state->weights.ln2_gamma[i]);
    hipFree(state->weights.ln2_beta[i]);
    hipFree(state->weights.mlp_fc1_weights[i]);
    hipFree(state->weights.mlp_fc1_bias[i]);
    hipFree(state->weights.mlp_fc2_weights[i]);
    hipFree(state->weights.mlp_fc2_bias[i]);

    // Activations
    hipFree(state->activations.layer_inputs[i]);
    hipFree(state->activations.queries[i]);
    hipFree(state->activations.keys[i]);
    hipFree(state->activations.values[i]);
    hipFree(state->activations.queries_reshaped[i]);
    hipFree(state->activations.keys_reshaped[i]);
    hipFree(state->activations.values_reshaped[i]);
    hipFree(state->activations.attention_scores[i]);
    hipFree(state->activations.attention_weights[i]);
    hipFree(state->activations.attention_output[i]);
    hipFree(state->activations.attention_proj[i]);
    hipFree(state->activations.post_attn[i]);
    hipFree(state->activations.ln1_outputs[i]);
    hipFree(state->activations.mlp_fc1[i]);
    hipFree(state->activations.mlp_gelu[i]);
    hipFree(state->activations.mlp_fc2[i]);
    hipFree(state->activations.post_mlp[i]);
    hipFree(state->activations.ln2_outputs[i]);
    hipFree(state->activations.query_input[i]);
    hipFree(state->activations.key_input[i]);
    hipFree(state->activations.value_input[i]);
    hipFree(state->activations.mlp_fc1_input[i]);

    // Gradient buffers (weights)
    hipFree(state->gradients.attention_query_weights[i]);
    hipFree(state->gradients.attention_key_weights[i]);
    hipFree(state->gradients.attention_value_weights[i]);
    hipFree(state->gradients.attention_output_weights[i]);
    hipFree(state->gradients.attention_query_bias[i]);
    hipFree(state->gradients.attention_key_bias[i]);
    hipFree(state->gradients.attention_value_bias[i]);
    hipFree(state->gradients.attention_output_bias[i]);
    hipFree(state->gradients.ln1_gamma[i]);
    hipFree(state->gradients.ln1_beta[i]);
    hipFree(state->gradients.ln2_gamma[i]);
    hipFree(state->gradients.ln2_beta[i]);
    hipFree(state->gradients.mlp_fc1_weights[i]);
    hipFree(state->gradients.mlp_fc1_bias[i]);
    hipFree(state->gradients.mlp_fc2_weights[i]);
    hipFree(state->gradients.mlp_fc2_bias[i]);

    hipFree(state->gradients.attention_query_weights[i]);
    hipFree(state->gradients.attention_key_weights[i]);
    hipFree(state->gradients.attention_value_weights[i]);
    hipFree(state->gradients.attention_output_weights[i]);
    hipFree(state->gradients.attention_query_bias[i]);
    hipFree(state->gradients.attention_key_bias[i]);
    hipFree(state->gradients.attention_value_bias[i]);
    hipFree(state->gradients.attention_output_bias[i]);
    hipFree(state->gradients.ln1_gamma[i]);
    hipFree(state->gradients.ln1_beta[i]);
    hipFree(state->gradients.ln2_gamma[i]);
    hipFree(state->gradients.ln2_beta[i]);
    hipFree(state->gradients.mlp_fc1_weights[i]);
    hipFree(state->gradients.mlp_fc1_bias[i]);
    hipFree(state->gradients.mlp_fc2_weights[i]);
    hipFree(state->gradients.mlp_fc2_bias[i]);

    // Gradient buffers (intermediates)
    hipFree(state->gradients.layer_inputs[i]);
    hipFree(state->gradients.queries[i]);
    hipFree(state->gradients.keys[i]);
    hipFree(state->gradients.values[i]);
    hipFree(state->gradients.query_input[i]);
    hipFree(state->gradients.key_input[i]);
    hipFree(state->gradients.value_input[i]);
    hipFree(state->gradients.attention_scores[i]);
    hipFree(state->gradients.attention_weights[i]);
    hipFree(state->gradients.attention_output[i]);
    hipFree(state->gradients.attention_proj[i]);
    hipFree(state->gradients.post_attn[i]);
    hipFree(state->gradients.ln1_outputs[i]);
    hipFree(state->gradients.mlp_fc1[i]);
    hipFree(state->gradients.mlp_fc1_input[i]);
    hipFree(state->gradients.mlp_gelu[i]);
    hipFree(state->gradients.mlp_fc2[i]);
    hipFree(state->gradients.post_mlp[i]);
    hipFree(state->gradients.ln2_outputs[i]);

    // Optimizer momentum
    hipFree(state->optimizer.momentum.attention_query_weights[i]);
    hipFree(state->optimizer.momentum.attention_key_weights[i]);
    hipFree(state->optimizer.momentum.attention_value_weights[i]);
    hipFree(state->optimizer.momentum.attention_output_weights[i]);
    hipFree(state->optimizer.momentum.attention_query_bias[i]);
    hipFree(state->optimizer.momentum.attention_key_bias[i]);
    hipFree(state->optimizer.momentum.attention_value_bias[i]);
    hipFree(state->optimizer.momentum.attention_output_bias[i]);
    hipFree(state->optimizer.momentum.ln1_gamma[i]);
    hipFree(state->optimizer.momentum.ln1_beta[i]);
    hipFree(state->optimizer.momentum.ln2_gamma[i]);
    hipFree(state->optimizer.momentum.ln2_beta[i]);
    hipFree(state->optimizer.momentum.mlp_fc1_weights[i]);
    hipFree(state->optimizer.momentum.mlp_fc1_bias[i]);
    hipFree(state->optimizer.momentum.mlp_fc2_weights[i]);
    hipFree(state->optimizer.momentum.mlp_fc2_bias[i]);

    // Optimizer velocity
    hipFree(state->optimizer.velocity.attention_query_weights[i]);
    hipFree(state->optimizer.velocity.attention_key_weights[i]);
    hipFree(state->optimizer.velocity.attention_value_weights[i]);
    hipFree(state->optimizer.velocity.attention_output_weights[i]);
    hipFree(state->optimizer.velocity.attention_query_bias[i]);
    hipFree(state->optimizer.velocity.attention_key_bias[i]);
    hipFree(state->optimizer.velocity.attention_value_bias[i]);
    hipFree(state->optimizer.velocity.attention_output_bias[i]);
    hipFree(state->optimizer.velocity.ln1_gamma[i]);
    hipFree(state->optimizer.velocity.ln1_beta[i]);
    hipFree(state->optimizer.velocity.ln2_gamma[i]);
    hipFree(state->optimizer.velocity.ln2_beta[i]);
    hipFree(state->optimizer.velocity.mlp_fc1_weights[i]);
    hipFree(state->optimizer.velocity.mlp_fc1_bias[i]);
    hipFree(state->optimizer.velocity.mlp_fc2_weights[i]);
    hipFree(state->optimizer.velocity.mlp_fc2_bias[i]);
  }

  // Free final layer
  hipFree(state->weights.final_ln_gamma);
  hipFree(state->weights.final_ln_beta);
  hipFree(state->weights.output_weights);

  hipFree(state->activations.embedded_tokens);
  hipFree(state->activations.final_ln_output);
  hipFree(state->activations.logits);
  hipFree(state->activations.softmax_output);
  hipFree(state->activations.loss);

  hipFree(state->gradients.token_embeddings);
  hipFree(state->gradients.final_ln_gamma);
  hipFree(state->gradients.final_ln_beta);
  hipFree(state->gradients.output_weights);
  hipFree(state->gradients.final_ln_output);
  hipFree(state->gradients.logits);

  hipFree(state->optimizer.momentum.token_embeddings);
  hipFree(state->optimizer.momentum.final_ln_gamma);
  hipFree(state->optimizer.momentum.final_ln_beta);
  hipFree(state->optimizer.momentum.output_weights);

  hipFree(state->optimizer.momentum.position_embeddings);
  hipFree(state->optimizer.velocity.position_embeddings);

  hipFree(state->optimizer.velocity.token_embeddings);
  hipFree(state->optimizer.velocity.final_ln_gamma);
  hipFree(state->optimizer.velocity.final_ln_beta);
  hipFree(state->optimizer.velocity.output_weights);

  // Delete host pointer arrays
  delete[] state->weights.attention_query_weights;
  delete[] state->weights.attention_key_weights;
  delete[] state->weights.attention_value_weights;
  delete[] state->weights.attention_output_weights;
  delete[] state->weights.attention_query_bias;
  delete[] state->weights.attention_key_bias;
  delete[] state->weights.attention_value_bias;
  delete[] state->weights.attention_output_bias;
  delete[] state->weights.ln1_gamma;
  delete[] state->weights.ln1_beta;
  delete[] state->weights.ln2_gamma;
  delete[] state->weights.ln2_beta;
  delete[] state->weights.mlp_fc1_weights;
  delete[] state->weights.mlp_fc1_bias;
  delete[] state->weights.mlp_fc2_weights;
  delete[] state->weights.mlp_fc2_bias;

  delete[] state->activations.layer_inputs;
  delete[] state->activations.queries;
  delete[] state->activations.keys;
  delete[] state->activations.values;
  delete[] state->activations.queries_reshaped;
  delete[] state->activations.keys_reshaped;
  delete[] state->activations.values_reshaped;
  delete[] state->activations.attention_scores;
  delete[] state->activations.attention_weights;
  delete[] state->activations.attention_output;
  delete[] state->activations.attention_proj;
  delete[] state->activations.post_attn;
  delete[] state->activations.ln1_outputs;
  delete[] state->activations.mlp_fc1;
  delete[] state->activations.mlp_gelu;
  delete[] state->activations.mlp_fc2;
  delete[] state->activations.post_mlp;
  delete[] state->activations.ln2_outputs;
  delete[] state->activations.query_input;
  delete[] state->activations.key_input;
  delete[] state->activations.value_input;
  delete[] state->activations.mlp_fc1_input;

  delete[] state->gradients.attention_query_weights;
  delete[] state->gradients.attention_key_weights;
  delete[] state->gradients.attention_value_weights;
  delete[] state->gradients.attention_output_weights;
  delete[] state->gradients.attention_query_bias;
  delete[] state->gradients.attention_key_bias;
  delete[] state->gradients.attention_value_bias;
  delete[] state->gradients.attention_output_bias;
  delete[] state->gradients.ln1_gamma;
  delete[] state->gradients.ln1_beta;
  delete[] state->gradients.ln2_gamma;
  delete[] state->gradients.ln2_beta;
  delete[] state->gradients.mlp_fc1_weights;
  delete[] state->gradients.mlp_fc1_bias;
  delete[] state->gradients.mlp_fc2_weights;
  delete[] state->gradients.mlp_fc2_bias;
  delete[] state->gradients.layer_inputs;
  delete[] state->gradients.queries;
  delete[] state->gradients.keys;
  delete[] state->gradients.values;
  delete[] state->gradients.query_input;
  delete[] state->gradients.key_input;
  delete[] state->gradients.value_input;
  delete[] state->gradients.attention_scores;
  delete[] state->gradients.attention_weights;
  delete[] state->gradients.attention_output;
  delete[] state->gradients.attention_proj;
  delete[] state->gradients.post_attn;
  delete[] state->gradients.ln1_outputs;
  delete[] state->gradients.mlp_fc1;
  delete[] state->gradients.mlp_fc1_input;
  delete[] state->gradients.mlp_gelu;
  delete[] state->gradients.mlp_fc2;
  delete[] state->gradients.post_mlp;
  delete[] state->gradients.ln2_outputs;

  delete[] state->optimizer.momentum.attention_query_weights;
  delete[] state->optimizer.momentum.attention_key_weights;
  delete[] state->optimizer.momentum.attention_value_weights;
  delete[] state->optimizer.momentum.attention_output_weights;
  delete[] state->optimizer.momentum.attention_query_bias;
  delete[] state->optimizer.momentum.attention_key_bias;
  delete[] state->optimizer.momentum.attention_value_bias;
  delete[] state->optimizer.momentum.attention_output_bias;
  delete[] state->optimizer.momentum.ln1_gamma;
  delete[] state->optimizer.momentum.ln1_beta;
  delete[] state->optimizer.momentum.ln2_gamma;
  delete[] state->optimizer.momentum.ln2_beta;
  delete[] state->optimizer.momentum.mlp_fc1_weights;
  delete[] state->optimizer.momentum.mlp_fc1_bias;
  delete[] state->optimizer.momentum.mlp_fc2_weights;
  delete[] state->optimizer.momentum.mlp_fc2_bias;

  delete[] state->optimizer.velocity.attention_query_weights;
  delete[] state->optimizer.velocity.attention_key_weights;
  delete[] state->optimizer.velocity.attention_value_weights;
  delete[] state->optimizer.velocity.attention_output_weights;
  delete[] state->optimizer.velocity.attention_query_bias;
  delete[] state->optimizer.velocity.attention_key_bias;
  delete[] state->optimizer.velocity.attention_value_bias;
  delete[] state->optimizer.velocity.attention_output_bias;
  delete[] state->optimizer.velocity.ln1_gamma;
  delete[] state->optimizer.velocity.ln1_beta;
  delete[] state->optimizer.velocity.ln2_gamma;
  delete[] state->optimizer.velocity.ln2_beta;
  delete[] state->optimizer.velocity.mlp_fc1_weights;
  delete[] state->optimizer.velocity.mlp_fc1_bias;
  delete[] state->optimizer.velocity.mlp_fc2_weights;
  delete[] state->optimizer.velocity.mlp_fc2_bias;

  std::cout << "Model freed" << std::endl;
}

void save_checkpoint(const TrainingState* state, const std::string& filepath, int epoch, float loss){
  std:: cout << "saving checkpoint to " << filepath << "..." << '\n';

  std::ofstream file(filepath, std::ios::binary);
  if(!file){
    std::cerr << "failed to open checkpoint file for writing" << '\n';
    return;
  }

  file.write(reinterpret_cast<const char*>(&state->config.vocab_size), sizeof(int));
  file.write(reinterpret_cast<const char*>(&state->config.embed_dim), sizeof(int));
  file.write(reinterpret_cast<const char*>(&state->config.num_layers), sizeof(int));
  file.write(reinterpret_cast<const char*>(&state->config.num_heads), sizeof(int));
  file.write(reinterpret_cast<const char*>(&state->config.seq_len), sizeof(int));
  file.write(reinterpret_cast<const char*>(&state->config.batch_size), sizeof(int));
  file.write(reinterpret_cast<const char*>(&epoch), sizeof(int));
  file.write(reinterpret_cast<const char*>(&loss), sizeof(float));

  int vocab_size = state->config.vocab_size;
  int embed_dim = state->config.embed_dim;
  int num_layers = state->config.num_layers;
  int seq_len = state->config.seq_len;
  int mlp_hidden = 4 * embed_dim;

  auto save_weights = [&](float* d_weights, size_t size){
    std::vector<float> h_weights(size);
    hipMemcpy(h_weights.data(), d_weights, size * sizeof(float), hipMemcpyDeviceToHost);
    file.write(reinterpret_cast<const char*>(h_weights.data()), size * sizeof(float));
  };

  // Save embeddings
  save_weights(state->weights.token_embeddings, vocab_size * embed_dim);
  save_weights(state->weights.position_embeddings, seq_len * embed_dim);

  // Save per-layer weights
  for (int i = 0; i < num_layers; i++) {
    save_weights(state->weights.attention_query_weights[i], embed_dim * embed_dim);
    save_weights(state->weights.attention_key_weights[i], embed_dim * embed_dim);
    save_weights(state->weights.attention_value_weights[i], embed_dim * embed_dim);
    save_weights(state->weights.attention_output_weights[i], embed_dim * embed_dim);
    
    save_weights(state->weights.attention_query_bias[i], embed_dim);
    save_weights(state->weights.attention_key_bias[i], embed_dim);
    save_weights(state->weights.attention_value_bias[i], embed_dim);
    save_weights(state->weights.attention_output_bias[i], embed_dim);
    
    save_weights(state->weights.ln1_gamma[i], embed_dim);
    save_weights(state->weights.ln1_beta[i], embed_dim);
    save_weights(state->weights.ln2_gamma[i], embed_dim);
    save_weights(state->weights.ln2_beta[i], embed_dim);
    
    save_weights(state->weights.mlp_fc1_weights[i], embed_dim * mlp_hidden);
    save_weights(state->weights.mlp_fc1_bias[i], mlp_hidden);
    save_weights(state->weights.mlp_fc2_weights[i], mlp_hidden * embed_dim);
    save_weights(state->weights.mlp_fc2_bias[i], embed_dim);
  }

  // Save final layer
  save_weights(state->weights.final_ln_gamma, embed_dim);
  save_weights(state->weights.final_ln_beta, embed_dim);
  save_weights(state->weights.output_weights, embed_dim * vocab_size);

  file.close();
  std::cout << "Checkpoint saved" << std::endl;
}

void load_checkpoint(TrainingState* state, const std::string& filepath){
  std::cout << "loading checkpoint from " << filepath << "..." << '\n';

  std::ifstream file(filepath, std::ios::binary);
  if(!file){
    std::cerr << "failed to open checkpoint file for reading" << '\n';
    return;
  }

  // Read config and metadata
  ModelConfig saved_config;
  int saved_epoch;
  float saved_loss;
  
  file.read(reinterpret_cast<char*>(&saved_config.vocab_size), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_config.embed_dim), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_config.num_layers), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_config.num_heads), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_config.seq_len), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_config.batch_size), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_epoch), sizeof(int));
  file.read(reinterpret_cast<char*>(&saved_loss), sizeof(float));

  // Verify config matches
  if (saved_config.vocab_size != state->config.vocab_size ||
      saved_config.embed_dim != state->config.embed_dim ||
      saved_config.num_layers != state->config.num_layers ||
      saved_config.num_heads != state->config.num_heads) {
    std::cerr << "Error: Checkpoint config doesn't match current model config!" << std::endl;
    std::cerr << "Checkpoint: vocab=" << saved_config.vocab_size 
              << " embed=" << saved_config.embed_dim
              << " layers=" << saved_config.num_layers 
              << " heads=" << saved_config.num_heads << std::endl;
    std::cerr << "Current: vocab=" << state->config.vocab_size 
              << " embed=" << state->config.embed_dim
              << " layers=" << state->config.num_layers 
              << " heads=" << state->config.num_heads << std::endl;
    file.close();
    return;
  }

  std::cout << "Loading from epoch " << saved_epoch << " (loss: " << saved_loss << ")" << std::endl;

  int vocab_size = state->config.vocab_size;
  int embed_dim = state->config.embed_dim;
  int num_layers = state->config.num_layers;
  int seq_len = state->config.seq_len;
  int mlp_hidden = 4 * embed_dim;

  // Helper to read from file and copy to device
  auto load_weights = [&](float* d_weights, size_t size) {
    std::vector<float> h_weights(size);
    file.read(reinterpret_cast<char*>(h_weights.data()), size * sizeof(float));
    hipMemcpy(d_weights, h_weights.data(), size * sizeof(float), hipMemcpyHostToDevice);
  };

  // Load embeddings
  load_weights(state->weights.token_embeddings, vocab_size * embed_dim);
  load_weights(state->weights.position_embeddings, seq_len * embed_dim);

  // Load per-layer weights
  for (int i = 0; i < num_layers; i++) {
    load_weights(state->weights.attention_query_weights[i], embed_dim * embed_dim);
    load_weights(state->weights.attention_key_weights[i], embed_dim * embed_dim);
    load_weights(state->weights.attention_value_weights[i], embed_dim * embed_dim);
    load_weights(state->weights.attention_output_weights[i], embed_dim * embed_dim);
    
    load_weights(state->weights.attention_query_bias[i], embed_dim);
    load_weights(state->weights.attention_key_bias[i], embed_dim);
    load_weights(state->weights.attention_value_bias[i], embed_dim);
    load_weights(state->weights.attention_output_bias[i], embed_dim);
    
    load_weights(state->weights.ln1_gamma[i], embed_dim);
    load_weights(state->weights.ln1_beta[i], embed_dim);
    load_weights(state->weights.ln2_gamma[i], embed_dim);
    load_weights(state->weights.ln2_beta[i], embed_dim);
    
    load_weights(state->weights.mlp_fc1_weights[i], embed_dim * mlp_hidden);
    load_weights(state->weights.mlp_fc1_bias[i], mlp_hidden);
    load_weights(state->weights.mlp_fc2_weights[i], mlp_hidden * embed_dim);
    load_weights(state->weights.mlp_fc2_bias[i], embed_dim);
  }

  // Load final layer
  load_weights(state->weights.final_ln_gamma, embed_dim);
  load_weights(state->weights.final_ln_beta, embed_dim);
  load_weights(state->weights.output_weights, embed_dim * vocab_size);

  file.close();
  std::cout << "Checkpoint loaded successfully" << std::endl;
}

float get_learning_rate(int step, int warmup_steps, int total_steps, float max_lr){
  // DIAGNOSTIC: constant LR to test if warmup is causing divergence
  return 5e-5f;

  // Original schedule (disabled for testing):
  // if(step < warmup_steps){
  //   return max_lr * (float)(step + 1) / warmup_steps;
  // }else{
  //   float progress = (float)(step - warmup_steps) / (total_steps - warmup_steps);
  //   float min_lr = max_lr * 0.1f;
  //   return min_lr + (max_lr - min_lr) * 0.5f * (1.0f + cosf(3.14159f * progress));
  // }
}

}//namespace training
