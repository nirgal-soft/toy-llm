#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "training.h"
#include "kernels.cuh"
#include "json.hpp"

using json = nlohmann::json;

// Load vocabulary
struct Vocabulary {
  std::vector<std::string> idx_to_token;
  int vocab_size;
};

Vocabulary load_vocabulary(const std::string& vocab_path) {
  Vocabulary vocab;
  std::ifstream file(vocab_path);
  if (!file) {
    std::cerr << "Failed to open vocab file: " << vocab_path << std::endl;
    exit(1);
  }

  // Parse JSON
  json j;
  file >> j;
  file.close();

  // Get vocab size
  vocab.vocab_size = j["vocab_size"];
  vocab.idx_to_token.resize(vocab.vocab_size);

  std::cout << "Reading vocab with " << vocab.vocab_size << " tokens..." << std::endl;

  // Build idx_to_token array from vocab mapping
  for (auto& [token, id] : j["vocab"].items()) {
    int token_id = id.get<int>();
    if (token_id >= 0 && token_id < vocab.vocab_size) {
      vocab.idx_to_token[token_id] = token;
    }
  }

  std::cout << "Loaded vocabulary with " << vocab.vocab_size << " tokens" << std::endl;
  return vocab;
}

// Simple tokenization (splits on whitespace and matches vocab)
std::vector<int> tokenize(const std::string& text, const Vocabulary& vocab) {
  std::vector<int> tokens;
  std::string current_token;
  
  for (char c : text) {
    if (c == ' ' || c == '\n' || c == '\t') {
      if (!current_token.empty()) {
        // Try to find token in vocab
        bool found = false;
        for (int i = 0; i < vocab.vocab_size; i++) {
          if (vocab.idx_to_token[i] == current_token) {
            tokens.push_back(i);
            found = true;
            break;
          }
        }
        if (!found) {
          // Use token 0 (unknown) if not found
          tokens.push_back(0);
        }
        current_token.clear();
      }
    } else {
      current_token += c;
    }
  }
  
  // Handle last token
  if (!current_token.empty()) {
    bool found = false;
    for (int i = 0; i < vocab.vocab_size; i++) {
      if (vocab.idx_to_token[i] == current_token) {
        tokens.push_back(i);
        found = true;
        break;
      }
    }
    if (!found) {
      tokens.push_back(0);
    }
  }
  
  return tokens;
}

// Sample token from logits (CUDA kernel)
__global__ void sample_token_kernel(float* logits, int* output, int vocab_size,
                                    float temperature, unsigned long long seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx != 0) return;

  hiprandState state;
  hiprand_init(seed, 0, 0, &state);

  // Find max for numerical stability
  float max_logit = logits[0];
  for (int i = 1; i < vocab_size; i++) {
    max_logit = fmaxf(max_logit, logits[i]);
  }

  // Apply temperature and softmax
  float sum = 0.0f;
  for (int i = 0; i < vocab_size; i++) {
    logits[i] = expf((logits[i] - max_logit) / temperature);
    sum += logits[i];
  }

  // Normalize
  for (int i = 0; i < vocab_size; i++) {
    logits[i] /= sum;
  }

  // Sample
  float rand_val = hiprand_uniform(&state);
  float cumsum = 0.0f;
  for (int i = 0; i < vocab_size; i++) {
    cumsum += logits[i];
    if (rand_val <= cumsum) {
      *output = i;
      return;
    }
  }
  *output = vocab_size - 1;
}

// Generate text
void generate(training::TrainingState* state, const Vocabulary& vocab,
              const std::vector<int>& prompt_tokens, int max_new_tokens,
              float temperature) {
  
  int seq_len = state->config.seq_len;
  int vocab_size = state->config.vocab_size;
  
  if (prompt_tokens.size() >= static_cast<size_t>(seq_len)) {
    std::cerr << "Prompt too long (max " << seq_len - 1 << " tokens)" << std::endl;
    return;
  }

  // Device memory
  int* d_tokens;
  int* d_next_token;
  float* d_last_logits;
  
  hipMalloc(&d_tokens, seq_len * sizeof(int));
  hipMalloc(&d_next_token, sizeof(int));
  hipMalloc(&d_last_logits, vocab_size * sizeof(float));
  
  // Initialize context with prompt
  std::vector<int> context(seq_len, 0);
  for (size_t i = 0; i < prompt_tokens.size(); i++) {
    context[i] = prompt_tokens[i];
  }
  hipMemcpy(d_tokens, context.data(), seq_len * sizeof(int), hipMemcpyHostToDevice);
  
  int current_len = prompt_tokens.size();
  
  std::cout << "\nPrompt: ";
  for (int token : prompt_tokens) {
    std::cout << vocab.idx_to_token[token] << " ";
  }
  std::cout << "\n\nGenerated: " << std::flush;
  
  for (int step = 0; step < max_new_tokens && current_len < seq_len; step++) {
    // Run forward pass (batch_size=1)
    training::forward_pass(state, d_tokens, 1, seq_len);
    
    // Extract logits for last generated position
    int last_pos = current_len - 1;
    hipMemcpy(d_last_logits,
               state->activations.logits + last_pos * vocab_size,
               vocab_size * sizeof(float),
               hipMemcpyDeviceToDevice);
    
    // Sample next token
    sample_token_kernel<<<1, 1>>>(d_last_logits, d_next_token, vocab_size,
                                   temperature, time(NULL) + step);
    hipDeviceSynchronize();
    
    // Get token
    int next_token;
    hipMemcpy(&next_token, d_next_token, sizeof(int), hipMemcpyDeviceToHost);
    
    // Print
    std::cout << vocab.idx_to_token[next_token] << " " << std::flush;
    
    // Update context
    context[current_len] = next_token;
    hipMemcpy(d_tokens + current_len, &next_token, sizeof(int), hipMemcpyHostToDevice);
    current_len++;
  }
  
  std::cout << "\n" << std::endl;
  
  hipFree(d_tokens);
  hipFree(d_next_token);
  hipFree(d_last_logits);
}

int main(int argc, char** argv) {
  // if (argc < 4) {
  //   std::cout << "Usage: " << argv[0] << " <checkpoint.bin> <vocab.bin> <prompt> [max_tokens] [temperature]" << std::endl;
  //   std::cout << "Example: " << argv[0] << " checkpoints/model_batch_2500.bin data/preprocessed/vocab.bin \"Once upon a time\" 50 0.8" << std::endl;
  //   return 1;
  // }

  // std::string checkpoint_path = argv[1];
  // std::string vocab_path = argv[2];
  std::string checkpoint_path = "./model_batch_45001.bin";
  std::string vocab_path = "./data/preprocessed/vocab.json";
  std::string prompt_text = (argc > 1) ? argv[1] : "Once upon a time";
  int max_tokens = (argc > 2) ? std::stoi(argv[2]) : 50;
  float temperature = (argc > 3) ? std::stof(argv[3]) : 0.3f;

  std::cout << "=== Text Generation ===" << std::endl;
  std::cout << "Checkpoint: " << checkpoint_path << std::endl;
  std::cout << "Prompt: " << prompt_text << std::endl;
  std::cout << "Max tokens: " << max_tokens << std::endl;
  std::cout << "Temperature: " << temperature << "\n" << std::endl;

  // Load vocabulary
  Vocabulary vocab = load_vocabulary(vocab_path);
  std::cout << "Loaded vocabulary with " << vocab.vocab_size << " tokens" << std::endl;

  // Create config (will be overwritten by checkpoint)
  training::ModelConfig config;
  config.vocab_size = vocab.vocab_size;
  config.embed_dim = 128;  // Will be overwritten
  config.num_layers = 3;    // Will be overwritten
  config.num_heads = 8;     // Will be overwritten
  config.seq_len = 64;      // Will be overwritten
  config.batch_size = 1;    // For inference

  // Allocate model
  training::TrainingState state;
  state.config = config;
  training::allocate_model(&state, config);
  std::cout << "Allocated model" << std::endl;

  // Load checkpoint (this will verify config matches)
  training::load_checkpoint(&state, checkpoint_path);
  std::cout << "Loaded checkpoint" << std::endl;

  // Tokenize prompt
  std::vector<int> prompt_tokens = tokenize(prompt_text, vocab);
  std::cout << "Tokenized prompt into " << prompt_tokens.size() << " tokens" << std::endl;

  // Generate
  generate(&state, vocab, prompt_tokens, max_tokens, temperature);

  std::cout << "=== Generation Complete ===" << std::endl;

  // Cleanup
  training::free_model(&state);

  return 0;
}
